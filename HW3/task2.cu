
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void task2(int *dA){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	dA[idx] = threadIdx.x + blockIdx.x;
}

int main(){
	int *dA;
       	int *hA = new int[16];
	int size = 16*sizeof(int);

	hipMalloc((void **)&dA, size);
	task2<<<2,8>>>(dA);
	hipDeviceSynchronize();
	hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);

	for(int i = 1; i < 16; i++) {
		cout << hA[i] << " ";
	}
	cout << endl;

	hipFree(dA);
	delete[] hA;
return 0;
}
