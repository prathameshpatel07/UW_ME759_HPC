#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int row_idx = idx/n;
	unsigned int col_idx = idx%n;
	C[idx] = 0;
	if(idx < n*n) {
	 	for(unsigned int k = 0; k < n; k++) {
			C[idx] += A[n*row_idx + k] * B[n*k + col_idx];
		}	
	}
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {
	unsigned int m = threads_per_block;
	matmul_kernel<<<(n*n + m-1)/m, m>>>(A, B, C, n);
	hipDeviceSynchronize();
}
