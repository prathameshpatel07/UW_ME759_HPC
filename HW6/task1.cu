#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "mmul.h"

using namespace std;

int main(int argc, char** argv){
	float *A, *B, *C;
//	float *dC;
	unsigned int n = atoi(argv[1]);
	unsigned int n_tests = atoi(argv[2]);
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipMallocManaged((void **)&A, n*n*sizeof(float));
	hipMallocManaged((void **)&B, n*n*sizeof(float));
	hipMallocManaged((void **)&C, n*n*sizeof(float));
	//hipMalloc((void **)&dC, n*n*sizeof(float));

	for(unsigned int i = 0; i < n; i++) {
		for(unsigned j = 0; j < n; j++) {
		A[n*j + i] = (float) ((i == j)? 1.0 : 0); //Identity Matrix
		B[n*j + i] = (float) n*i + j; 		 //Unique index matrix
		C[n*j + i] = 0;
		}
	}
	//hipblasSetMatrix(n, n, sizeof(float), A, n, B, n);
	//hipblasGetMatrix(n, n, sizeof(float), C, n, dC, n);
	hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	float ms_sum = 0;
	for(int i = 0; i < n_tests; i++) {
	float ms = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	mmul(handle, A, B, C, n);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	ms_sum += ms;
	}
	//Print template
	cout << ms_sum/n_tests << endl;

	hipblasDestroy(handle);
	hipFree(A);
	hipFree(B);
	hipFree(C);
return 0;
}
