#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matmul_kernel(const float* A, const float* B, float* C, unsigned int n) {

	int blocksize = blockDim.x;
	extern __shared__ float shared_arr[];
	float *As = shared_arr;
	float *Bs = (float*)&As[blocksize*blocksize];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int aBegin = by * blocksize * n;
	int aEnd = aBegin + n-1;
	int aStep = blocksize;
	int arow = aBegin/n + ty;
	int bBegin = bx * blocksize;
	int bStep = n * blocksize;
	int bcol = bBegin + tx;
	int c = n * blocksize * by + blocksize * bx;
	float Csub = 0;

	for (int a = aBegin, b = bBegin;a <= aEnd;a += aStep, b += bStep) {
		As[blocksize*ty + tx] = ((arow < n) && (a + tx <= aEnd))? A[a + n * ty + tx] : 0;
		Bs[blocksize*ty + tx] = ((bcol < n) && (a + tx <= aEnd))? B[b + n * ty + tx] : 0; 
		//Using aEnd condition for zero padding B matrix since column and row dimension of A and B respectively needs to be the same
		__syncthreads();
		for (int k = 0; k < blocksize; ++k)
			Csub += As[blocksize*ty + k] * Bs[blocksize*k + tx];
           	__syncthreads();
	}
	if((by*blocksize + ty < n) && (bx*blocksize + tx < n)) {
		C[c + n * ty + tx] = Csub;
	}
}

__host__ void matmul(const float* A, const float* B, float* C, unsigned int n, unsigned int block_dim) {
	dim3 dimBlock(block_dim, block_dim);
	dim3 dimGrid( (n + block_dim-1)/block_dim , (n + block_dim-1)/block_dim );
	size_t shared_array_size = (2*block_dim*block_dim)*sizeof(float);
	
	matmul_kernel<<<dimGrid, dimBlock, shared_array_size>>>(A, B, C, n);
	hipDeviceSynchronize();
}
