#include <iostream>
#include <hip/hip_runtime.h>
#include "scan.cuh"

using namespace std;

int main(int argc, char** argv){
	float *in, *out;
	unsigned int n = atoi(argv[1]);
	unsigned int threads_per_block = 1024;
	in = new float[n];
	out = new float[n];
	for(unsigned int i = 0; i < n; i++) {
		in[i] = 1;
	}

	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	scan(in, out, n, threads_per_block);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	//Print template
	cout << out[n-1] << endl;
	cout << ms << endl;

	delete[] in;
	delete[] out;
return 0;
}
