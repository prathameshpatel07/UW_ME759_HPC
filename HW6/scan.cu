#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void hillis_steele(float *g_odata, float *g_idata, float *blocksum, int n) {
    	extern volatile __shared__  float temp[]; // allocated on invocation
    	int pout = 0, pin = 1;
	int idx  = blockIdx.x*blockDim.x + threadIdx.x;
    	int thid = threadIdx.x;
	int m = blockDim.x;
	temp[thid] = (idx == 0 || idx >= n) ? 0: g_idata[idx-1];
	__syncthreads();
        for( int offset = 1; offset<m; offset *= 2 ) {
	        pout = 1 - pout; // swap double buffer indices
	        pin  = 1 - pout;
	        if (thid >= offset)
	            temp[pout*m+thid] = temp[pin*m+thid] + temp[pin*m+thid - offset];
		else
		    temp[pout*m+thid] = temp[pin*m+thid];
	        __syncthreads(); // I need this here before I start next iteration 
	}
	g_odata[idx] = temp[pout*m+thid]; // write output
	if(thid == blockDim.x -1 || idx == n-1) 
		blocksum[blockIdx.x] = temp[pout*m+thid];
}

__global__ void sum_kernel(float *g_odata, float *blocksum, int n) {
	int idx  = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n) {
		g_odata[idx] += blocksum[blockIdx.x];
	}
}
__host__ void scan(const float* in, float* out, unsigned int n, unsigned int threads_per_block) {
	float *g_idata, *g_odata;
	float *blocksum_in, *blocksum_out, *dummy;
	unsigned int m = threads_per_block;
	size_t shared_array_size = 2*m*sizeof(float);
	int blockdim = (n + m-1)/m;
	//float *blocksum = new float[blockdim];
	
	hipMalloc((void **)&g_idata, n*sizeof(float));
	hipMalloc((void **)&g_odata, n*sizeof(float));
	hipMallocManaged((void **)&blocksum_in, blockdim*sizeof(float));
	//cudaMalloc((void **)&blocksum_out, blockdim*sizeof(float));
	hipMallocManaged((void **)&blocksum_out, blockdim*sizeof(float));
	hipMallocManaged((void **)&dummy, 1*sizeof(float));
	hipMemcpy(g_idata, in, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(g_odata, 0, n*sizeof(float));
		
	hillis_steele<<<blockdim, m, shared_array_size>>>(g_odata, g_idata, blocksum_in, n);
	hipDeviceSynchronize();

	hillis_steele<<<1, m, shared_array_size>>>(blocksum_out, blocksum_in, dummy, blockdim);
	hipDeviceSynchronize();
	
	sum_kernel<<<blockdim, m>>>(g_odata, blocksum_out, n);
	hipDeviceSynchronize();

	hipMemcpy(out, g_odata, n*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(g_idata);
	hipFree(g_odata);
}
