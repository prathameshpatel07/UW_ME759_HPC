#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

using namespace std;

int main(int argc, char** argv){
	float *A, *B, *C;
	unsigned int n = atoi(argv[1]);
	unsigned int nthreads_perblock = atoi(argv[2]);

	hipMallocManaged((void **)&A, n*n*sizeof(float));
	hipMallocManaged((void **)&B, n*n*sizeof(float));
	hipMallocManaged((void **)&C, n*n*sizeof(float));

	for(unsigned int i = 0; i < n; i++) {
		for(unsigned j = 0; j < n; j++) {
		A[n*i + j] = (float) ((i == j)? 1.0 : 0); //Identity Matrix
		B[n*i + j] = (float) n*i + j; 		 //Unique index matrix
		}
	}

	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	matmul(A, B, C, n, nthreads_perblock);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	//Print template
	cout << C[n*n-1] << endl;
	cout << ms << endl;

	hipFree(A);
	hipFree(B);
	hipFree(C);
return 0;
}
