#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
	extern __shared__ float shared_arr[];
	float *msk = shared_arr;
	float *out = (float*)&msk[2*R+1];
	float *img = (float*)&out[blockDim.x];

	//Read Input elements
	int gidx = threadIdx.x + blockIdx.x * blockDim.x;
	int lidx = threadIdx.x + R;

	img[lidx] = image[gidx];
	if(threadIdx.x < R) {
		if((gidx - (signed) R) < 0) { //signed added to remove warnings
			img[lidx-R] = 0;
			img[lidx + blockDim.x] = image[gidx + blockDim.x];
		}
		else if(gidx + blockDim.x >= n) { 
			img[lidx - R] = image[gidx - R];
			img[lidx + blockDim.x] = 0;
		}
		else {
		img[lidx - R] = image[gidx - R];
		img[lidx + blockDim.x] = image[gidx + blockDim.x];
		}
	}

	msk[threadIdx.x] = mask[threadIdx.x];
	out[threadIdx.x] = 0;
	__syncthreads();

	//Applying stencil function
	for(int j = 0; j <= 2*R; j++) {
		  out[threadIdx.x] += img[threadIdx.x+j] * mask[j];
	}
	output[gidx] = out[threadIdx.x];
}

__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block) {

	unsigned int m = threads_per_block;
	size_t shared_array_size = (2*m + 4*R + 1)*sizeof(float); //Image=m+2R, Mask=2R+1, Output=m
	stencil_kernel<<<(n + m-1)/m, m, shared_array_size>>>(image, mask, output, n, R);
	hipDeviceSynchronize();
}
