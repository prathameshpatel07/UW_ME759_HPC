#include "count.cuh"
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
using namespace thrust;
void count(const device_vector<int> &d_in, device_vector<int> &values,
           device_vector<int> &counts) {
  int n = d_in.end() - d_in.begin();
  device_vector<int> in(n);
  copy(d_in.begin(), d_in.end(), in.begin());
  sort(in.begin(), in.end());
  int unique_entries = inner_product(in.begin(), in.end() - 1, in.begin() + 1,
                                     1, plus<int>(), not_equal_to<int>());
  values.resize(unique_entries);
  counts.resize(unique_entries);
  reduce_by_key(in.begin(), in.end(), constant_iterator<int>(1), values.begin(),
                counts.begin());
}
