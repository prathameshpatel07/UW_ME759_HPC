#include "hipblas.h"
#include "hip/hip_runtime.h"
void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
	const float one = 1;
	const float zero = 0;
	const float *alpha = &one;
	const float *beta = &zero;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, n, B, n, beta, C, n);	
	hipDeviceSynchronize();
}
