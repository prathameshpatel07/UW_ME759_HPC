#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void reduce_kernel(const int* g_idata, int* g_odata, unsigned int n) {
	extern __shared__ int shared_arr[];
	int *sdata = shared_arr;
	unsigned int idx  = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tidx = threadIdx.x;
	if(idx < n) {
		sdata[tidx] = g_idata[idx];
	}
	else
		sdata[tidx] = 0;

        __syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	    if (tidx < s) {
	          sdata[tidx] += sdata[tidx + s];
	    }
	    __syncthreads();
	}
    	if (tidx == 0) g_odata[blockIdx.x] = sdata[0];
}

__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block) {
	int *g_idata, *g_odata;
	unsigned int m = threads_per_block;
	size_t shared_array_size = m*sizeof(int);
	int blockdim = (N + m-1)/m;
	int *sum = new int[1];
	
	hipMalloc((void **)&g_idata, N*sizeof(int));
	hipMalloc((void **)&g_odata, blockdim*sizeof(int));
	hipMemcpy(g_idata, arr, N*sizeof(int), hipMemcpyHostToDevice);
	unsigned int nexti = 0;
	for(unsigned int i = N; i > 1; i=(i+ m-1)/m) {
		nexti = (i + m-1)/m;
		hipMemset(g_odata, 0, blockdim*sizeof(int));
		
		reduce_kernel<<< blockdim, m, shared_array_size>>>(g_idata, g_odata, N);
		hipDeviceSynchronize();
		
		hipMemset(g_idata, 0, N*sizeof(int));
		if(nexti != 0) {
			hipMemcpy(g_idata, g_odata, nexti*sizeof(int), hipMemcpyDeviceToDevice);
		}
	}

	hipMemcpy(sum, g_odata, 1*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(g_idata);
	hipFree(g_odata);
	return *sum;
}
