#include <iostream>
#include "vadd.cuh"

using namespace std;

int main(int argc, char** argv){
	float *a, *b;
	float *da, *db;
	unsigned int n = atoi(argv[1]);
	unsigned int size = n*sizeof(float);
	int m = 512; //Threads per block

	a = new float[n];
	b = new float[n];

	for(unsigned int i = 0; i < n; i++) {
		a[i] = (float) i;
		b[i] = (float) 2 * i;
	}

	hipMalloc((void **)&da, size);
	hipMalloc((void **)&db, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);

	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	vadd<<<(n + m-1)/m, m>>>(da, db, n);

//`	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	hipMemcpy(b, db, size, hipMemcpyDeviceToHost);
	//Print template
	cout << ms/1000 << endl;
	cout << b[0] << endl;
	cout << b[n-1] << endl;

	hipFree(da);
	hipFree(db);
	delete[] a;
	delete[] b;
return 0;
}
