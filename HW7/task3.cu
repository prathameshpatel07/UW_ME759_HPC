#include "count.cuh"
#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
using namespace thrust;
int main(int argc, char **argv) {
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n = atoi(argv[1]);
  host_vector<int> h_in(n);
  host_vector<int> h_counts(n);
  host_vector<int> h_values(n);
  device_vector<int> d_in(n);
  device_vector<int> d_values(n);
  device_vector<int> d_counts(n);
  for (unsigned int i = 0; i < n; i++)
    h_in[i] = (static_cast<int>(rand()) / static_cast<int>(RAND_MAX / 10));
  copy(h_in.begin(), h_in.end(), d_in.begin());

  hipEventRecord(start);
  count(d_in, d_values, d_counts);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  copy(d_values.begin(), d_values.end(), h_values.begin());
  copy(d_counts.begin(), d_counts.end(), h_counts.begin());

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  int unique_n = d_values.end() - d_values.begin();
  std::cout << h_values[unique_n - 1] << std::endl;
  std::cout << h_counts[unique_n - 1] << std::endl;
  std::cout << ms << std::endl;
  return 0;
}
