#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <cub/device/device_scan.cuh>
#include <cub/util_allocator.cuh>
#include <stdio.h>
using namespace hipcub;
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

int main(int argc, char **argv) {
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n = atoi(argv[1]);
  float *h_in = new float[n];
  for (unsigned int i = 0; i < n; i++)
    h_in[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX / 2));

  // Set up device arrays
  float *d_in = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_in, sizeof(float) * n));
  // Initialize device input
  CubDebugExit(
      hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
  // Setup device output array
  float *d_out = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_out, sizeof(float) * n));
  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                        d_in, d_out, n));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
  hipEventRecord(start);
  // Do the actual scan operation
  CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                        d_in, d_out, n));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  float *h_out = new float[n];
  CubDebugExit(
      hipMemcpy(h_out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost));

  std::cout << h_out[n - 1] << std::endl;
  std::cout << ms << std::endl;
  // Cleanup
  if (d_in)
    CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_out)
    CubDebugExit(g_allocator.DeviceFree(d_out));
  if (d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
  return 0;
}
