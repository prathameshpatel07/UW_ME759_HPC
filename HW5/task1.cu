#include <iostream>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

using namespace std;

int main(int argc, char** argv){
	int *arr;
	unsigned int n = atoi(argv[1]);
	unsigned int threads_per_block = atoi(argv[2]);
	int sum = 0;

	arr = new int[n];
	for(unsigned int i = 0; i < n; i++) {
		arr[i] = 1;
	}

	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	sum = reduce(arr, n, threads_per_block);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	//Print template
	cout << sum << endl;
	cout << ms << endl;

	delete[] arr;
return 0;
}
