#include <iostream>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

using namespace std;

int main(int argc, char** argv){
	float *img, *mask, *out;
	unsigned int n = atoi(argv[1]);
	unsigned int R = atoi(argv[2]);
	unsigned int threads_per_block = atoi(argv[3]);

	hipMallocManaged((void **)&img, n*sizeof(float));
	hipMallocManaged((void **)&mask, (2*R+1)*sizeof(float));
	hipMallocManaged((void **)&out, n*sizeof(float));

	for(unsigned int i = 0; i < n; i++) {
		img[i] = (float) i;
	}
	for(unsigned int j = 0; j < 2*R+1; j++) {
		mask[j] = (j == R)? 1.0 : 0;
	}

	//Measure Time template
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	stencil(img, mask, out, n, R, threads_per_block);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	//Print template
	cout << out[n-1] << endl;
	cout << ms << endl;

	hipFree(img);
	hipFree(mask);
	hipFree(out);
return 0;
}
