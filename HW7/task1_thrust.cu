#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
using namespace thrust;
int main(int argc, char **argv) {
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n = atoi(argv[1]);
  host_vector<int> h_in(n);
  device_vector<int> d_in(n);
  for (unsigned int i = 0; i < n; i++)
    h_in[i] = (static_cast<int>(rand()) / static_cast<int>(RAND_MAX / 2));
  copy(h_in.begin(), h_in.end(), d_in.begin());
  hipEventRecord(start);
  int sum = reduce(d_in.begin(), d_in.end(), (int)0, plus<int>());
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  std::cout << sum << std::endl;
  std::cout << ms << std::endl;
  return 0;
}
