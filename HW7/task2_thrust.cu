#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
using namespace thrust;
int main(int argc, char **argv) {
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n = atoi(argv[1]);
  host_vector<float> h_in(n);
  host_vector<float> h_out(n);
  device_vector<float> d_in(n);
  device_vector<float> d_out(n);
  for (unsigned int i = 0; i < n; i++)
    h_in[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX / 2));
  copy(h_in.begin(), h_in.end(), d_in.begin());
  hipEventRecord(start);
  exclusive_scan(d_in.begin(), d_in.end(), d_out.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  copy(d_out.begin(), d_out.end(), h_out.begin());
  float ms;
  hipEventElapsedTime(&ms, start, stop);

  std::cout << h_out[n - 1] << std::endl;
  std::cout << ms << std::endl;
  return 0;
}
