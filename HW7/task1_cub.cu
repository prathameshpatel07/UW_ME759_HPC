#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>
#include <stdio.h>
using namespace hipcub;
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

int main(int argc, char **argv) {
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int n = atoi(argv[1]);
  int *h_in = new int[n];
  for (unsigned int i = 0; i < n; i++)
    h_in[i] = (static_cast<int>(rand()) / static_cast<int>(RAND_MAX / 2));

  // Set up device arrays
  int *d_in = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_in, sizeof(int) * n));
  // Initialize device input
  CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * n, hipMemcpyHostToDevice));
  // Setup device output array
  int *d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_sum, sizeof(int) * 1));
  // Request and allocate temporary storage
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CubDebugExit(
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  hipEventRecord(start);
  // Do the actual reduce operation
  CubDebugExit(
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);
  int gpu_sum;
  CubDebugExit(
      hipMemcpy(&gpu_sum, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));

  std::cout << gpu_sum << std::endl;
  std::cout << ms << std::endl;
  // Cleanup
  if (d_in)
    CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_sum)
    CubDebugExit(g_allocator.DeviceFree(d_sum));
  if (d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

  return 0;
}
